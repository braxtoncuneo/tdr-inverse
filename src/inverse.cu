#include "hip/hip_runtime.h"
#include "utils.h"
#include <cstdio>
#include <fstream>
#include <string>
#include <vector>

using namespace helpers;

const int MAX_BLOCK_SIZE = 1024;

// (c) Sharma 2013
__global__ void fixRow(matrix_t *matrix, int size, int rowId) {
  // the ith row of the matrix
  __shared__ matrix_t Ri[MAX_BLOCK_SIZE];
  // The diagonal element for ith row
  __shared__ matrix_t Aii;
  int colId = threadIdx.x;
  Ri[colId] = matrix[size * rowId + colId];
  Aii = matrix[size * rowId + rowId];

  // Pivot if row is singular
  int i = 0;
  while (Aii == 0.) {
    if (matrix[size * i + rowId] != 0) {
#ifdef DEBUG
      printf("1. matrix[%d][%d] += matrix[%d][%d]\n", rowId, colId, i, colId);
#endif
      Ri[colId] += matrix[size * i + colId];
      Aii = Ri[rowId];
      break;
    }
    i++;
  }
#ifdef DEBUG
  printf("1. matrix[%d][%d] = %f\n", rowId, colId, Ri[colId]);
#endif
  __syncthreads();
  // Divide the whole row by the diagonal element making sure it is not 0
  Ri[colId] = Ri[colId] / Aii;
  matrix[size * rowId + colId] = Ri[colId];
#ifdef DEBUG
  printf("2. matrix[%d][%d] /= %f = %f\n", rowId, colId, Aii, Ri[colId]);
#endif
}

// (c) Sharma 2013
__global__ void fixColumn(matrix_t *matrix, int size, int colId) {
  int i = threadIdx.x;
  int j = blockIdx.x;
  // The colId column
  __shared__ matrix_t col[MAX_BLOCK_SIZE];
  // The jth element of the colId row
  __shared__ matrix_t AColIdj;
  // The jth column
  __shared__ matrix_t colj[MAX_BLOCK_SIZE];
  col[i] = matrix[i * size + colId];
  if (col[i] != 0) {
    colj[i] = matrix[i * size + j];
    AColIdj = matrix[colId * size + j];
    if (i != colId) {
      colj[i] = colj[i] - AColIdj * col[i];
#ifdef DEBUG
      printf("3. matrix[%d][%d] -= %f * %f = %f\n", i, j, AColIdj, col[i],
             colj[i]);
#endif
    }
    matrix[i * size + j] = colj[i];
  }
}

int main(int argc, char *argv[]) {

  std::ifstream matrixFile;
  std::ifstream solnFile;
  get_args(argc, argv, matrixFile, solnFile);

  size_t rows, cols;
  std::vector<matrix_t> soln;
  readCSV(solnFile, soln, rows, cols);

  std::vector<matrix_t> data;
  readCSV(matrixFile, data, rows, cols);

#ifdef DEBUG
  printMatrix(data.data(), rows, cols);
#endif

  // Timing objects
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Convert matrix to augmented form
  std::vector<matrix_t> aug;
  size_t aug_cols = 2 * cols;
  matrixToAug(data, aug, rows, cols);

  matrix_t *data_gpu = copy_to_gpu<matrix_t>(aug.data(), rows * aug_cols);

  hipEventRecord(start);

  // Main program flow
  for (size_t j = 0; j < rows; j++) {
    fixRow<<<1, aug_cols>>>(data_gpu, aug_cols, j);
    auto_throw(hipDeviceSynchronize());

    fixColumn<<<aug_cols, rows>>>(data_gpu, aug_cols, j);
    auto_throw(hipDeviceSynchronize());
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float msec;
  hipEventElapsedTime(&msec, start, stop);

  printf("Runtime: %f\n", msec);

  copy_from_gpu<matrix_t>(aug.data(), data_gpu, rows * aug_cols);

  // Convert matrix from augmented form
  data.clear();
  augToMatrix(data, aug, rows, cols);

#ifdef DEBUG
  printMatrix(data.data(), rows, cols);
#endif

  long double error;
  long double mae;
  for (size_t i = 0; i < soln.size(); i++) {
    error = std::abs(data[i] - soln[i]) / std::max(std::abs(soln[i]), std::abs(data[i]));
    mae += std::abs(data[i] - soln[i]);
    if (!std::isfinite(data[i]) || error > 0.0001) {
      fprintf(stderr, "matrix[%zu][%zu] expected % E got % E Error: %LE\n", i / cols,
              i % cols, soln[i], data[i], error);
    }
  }
  mae /= soln.size();
  fprintf(stderr, "Mean Absolute Error: %LE\n", mae);
  // printMatrix(data.data(), rows, cols);

  return 0;
}

#include "hip/hip_runtime.h"
#include "harmonize.cpp"
#include "utils.h"
#include <cstdio>
#include <fstream>
#include <string>
#include <vector>

using namespace util;

__global__ void storeAij(matrix_t *matrix, int size, matrix_t *Aij, int colId) {
  int rowId = threadIdx.x;
  Aij[rowId] = matrix[size*rowId + colId];
#ifdef DEBUG
  printf("0. A[%d][%d] = %f\n", rowId, colId, Aij[rowId]);
#endif

  if (rowId == colId)
    matrix[size*rowId + colId] = 1.0;
  else
    matrix[size*rowId + colId] = 0.0;
}

//struct Inverse { matrix_t *matrix, size_t size, size_t loc };

struct FixRow;
struct FixCol;

struct FixRow {
  using Type = void(*)(size_t rowId, size_t colId);

  template<typename PROGRAM>
  __device__ static void eval(PROGRAM prog, size_t rowId, size_t colId) {
    size_t size = prog.device.size.row;
    matrix_t Ri  = prog.device.matrix[size*rowId + colId];
    matrix_t Aii = prog.device.Aij[rowId];

#ifdef DEBUG
    printf("1. matrix[%lu][%lu] = %f\n", rowId, colId, Ri);
#endif

    Ri /= Aii;
    prog.device.matrix[size*rowId + colId] = Ri;

#ifdef DEBUG
    printf("2. matrix[%lu][%lu] /= %f = %f\n", rowId, colId, Aii, Ri);
#endif

    for (size_t i = 0; i < prog.device.size.col; i++) {
      matrix_t col = prog.device.Aij[i];
      if (col != 0) {
        prog.template async<FixCol>(rowId, i, colId, col);
      }
    }
  }
};

struct FixCol {
  using Type = void(*)(size_t colId, size_t i, size_t j, matrix_t col);

  template<typename PROGRAM>
  __device__ static void eval(PROGRAM prog, size_t colId, size_t i, size_t j, matrix_t col) {
    size_t size = prog.device.size.row;
    matrix_t colj    = prog.device.matrix[i*size + j];
    matrix_t AColIdj = prog.device.matrix[colId*size + j];
    if (i != colId) {
      colj -= AColIdj * col;

#ifdef DEBUG
      printf("3. matrix[%lu][%lu] -= %f * %f = %f\n", i, j, AColIdj, col, colj);
#endif
    }
    prog.device.matrix[i*size + j] = colj;
  }
};

struct Size2D {
  size_t row;
  size_t col;
};

struct InverseState {
  size_t *j;
  Size2D size;
  matrix_t *matrix;
  matrix_t *Aij;
  iter::AtomicIter<unsigned int>* iterator;
};

struct InverseSpec {
  typedef OpUnion<FixRow,FixCol>       OpSet;
  typedef           InverseState DeviceState;

  static const size_t STASH_SIZE =   16;
  static const size_t FRAME_SIZE = 8191;
  static const size_t  POOL_SIZE = 8191;

  /*
  // Defines the initialization function for programs of type 'ProgType'. This function is called by
  // all threads in all work groups just prior to beginning normal execution. States are accessible
  // through the 'device', 'group', and 'thread' variables, just like when defining async functions.
  //
  // Here, we initialize the work iterator to iterate over a range of integers unique to the work
  // group, distributing the set of integers to process more or less evenly across the set of
  // work groups.
  */
  template<typename PROGRAM>
  __device__ static void initialize(PROGRAM prog){

  }

  /*
  // Defines a function for programs of type 'ProgType' which is called by all threads in all work
  // groups after it is determined that there are no more promises to be processed. To be clear:
  // you should not perform any async calls in here and expect them to be always evaluated, since
  // the program is wrapping up and there is a good chance that no work groups will notice the
  // promises before exiting.
  //
  // Because promises are persistant across execution calls, if you want to queue work for the next
  // execution call, you can check if the current executing work group is the final one to call the
  // finalize function and queue work then. This will guarantee that the queued work will only be
  // evaluated in the next exec call.
  */
  template<typename PROGRAM>
  __device__ static void finalize(PROGRAM prog){

  }

  /*
  // Defines the work making function for programs of type 'ProgType'. This function is called by
  // work groups whenever they notice that they are running out of work to perform. To indicate
  // that there is still more work to perform, return 'true'. To indicate that there is no more
  // work left for the work group to make, return 'false', at which point, the work group will no
  // longer call this function for the remainder of the execution run.
  */
  template<typename PROGRAM>
  __device__ static bool make_work(PROGRAM prog){

      size_t size = prog.device.size.row;

      unsigned int iter_step_length = size;

      iter::Iter<unsigned int> iter = prog.device.iterator->leap(iter_step_length);

      unsigned int index;
      while(iter.step(index)){
        size_t rowId = *(prog.device.j);
        size_t colId = index;
        prog.template async<FixRow>(rowId, colId);
      }

      // TODO: size/2 because aug_matrix
      //if (prog.device.iterator->done() && prog.device.j < (prog.device.size/2)) {
      //  prog.device.iterator->reset(0, prog.device.size);
      //  prog.device.j++;
      //}

      return ! prog.device.iterator->done();

  }

};

typedef  HarmonizeProgram < InverseSpec > ProgType;

int main(int argc, char *argv[]) {

  cli::ArgSet args(argc, argv);

  std::ifstream matrixFile;
  std::ifstream solnFile;
  get_args(argc, argv, matrixFile, solnFile);

  size_t rows, cols;
  std::vector<matrix_t> soln;
  readCSV(solnFile, soln, rows, cols);

  std::vector<matrix_t> data;
  readCSV(matrixFile, data, rows, cols);

#ifdef DEBUG
  printMatrix(data.data(), rows, cols);
#endif

  // Timing objects
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  InverseState ds;

  ds.size.row = cols;
  ds.size.col = rows;

  //host::DevBuf<matrix_t> Aij = host::DevBuf<matrix_t>(rows);
  //ds.Aij = Aij;
  hipMalloc(&(ds.Aij), rows * sizeof(matrix_t));
  host::check_error();

  host::DevBuf<matrix_t> data_gpu = host::DevBuf<matrix_t>(ds.size.row * ds.size.col);
  data_gpu << data;
  // Assign the address of the device-side buffer to the device state so that the program
  // can know where to put its output.
  ds.matrix = data_gpu;

  host::DevBuf<size_t> j = host::DevBuf<size_t>();
  j << 1;
  ds.j = j;

  hipEventRecord(start);

    iter::AtomicIter<unsigned int> host_iter(0,ds.size.row);
    host::DevBuf<iter::AtomicIter<unsigned int>> iterator;
    iterator << host_iter; // Without this we get an access exception even though we set it later
    ds.iterator = iterator;

    // Declare and instance of type 'ProgType' with an arena size of 2^(20) with a device state
    // initialized to the value of our declared device state struct. The arena size of a
    // program determines how much extra space it has to store work if it cannot store
    // everything inside shared memory. If you are *certain* that no work will spill into
    // main memory, you may get some performance benefits by seting the arena size to zero.
    ProgType::Instance instance = ProgType::Instance(0x10000,ds);
    hipDeviceSynchronize();
    host::check_error();

    // Initialize the instance using 32 work groups
    init<ProgType>(instance,32);
    hipDeviceSynchronize();
    host::check_error();

  for (size_t cj = 0; cj < ds.size.col; cj++) {
    j << cj; // Push current row to gpu

    // Reset iter
    iter::AtomicIter<unsigned int> host_iter(0,ds.size.row);
    iterator << host_iter;

    storeAij<<<1, rows>>>(ds.matrix, ds.size.row, ds.Aij, cj);
    hipDeviceSynchronize();
    host::check_error();

    // Execute the instance using 240 work groups, with each work group performing up to
    // 65536 promise executions per thread before halting. If all promises are exhausted
    // before this, the program exits early.
    exec<ProgType>(instance,240,65536);
    hipDeviceSynchronize();
    host::check_error();
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float msec;
  hipEventElapsedTime(&msec, start, stop);

  printf("Runtime: %f\n", msec);


  data.clear();
  data_gpu >> data;

#ifdef DEBUG
  printMatrix(data.data(), rows, cols);
#endif

  printError(data.data(), soln.data(), rows, cols);

  return 0;
}
